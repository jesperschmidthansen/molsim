#include "hip/hip_runtime.h"

#include "sepcudaintgr.h"


// Again, I have no clue why these function must be defined here
// Octave's file scope versus CUDA...

#ifdef OCTAVE

__inline__ __device__ float sep_cuda_dot(float4 a){
	
	return (a.x*a.x + a.y*a.y + a.z*a.z);
	
}


__global__ void oct_sep_cuda_sumenergies(float3 *totalsum, float4* dx, float4 *dv, float4 *df, 
										 float dt, float *epot, unsigned npart){

	int id = blockIdx.x*blockDim.x + threadIdx.x;
	__shared__ float3 sums;

	if ( threadIdx.x==0 ) {
		sums.x = sums.y = sums.z = 0.0f;
	}
	__syncthreads();

	if ( id < npart ){
		float4 vel; 
		vel.x =  dv[id].x - 0.5*dt*df[id].x/dx[id].w;
		vel.y =  dv[id].y - 0.5*dt*df[id].y/dx[id].w;
		vel.z =  dv[id].z - 0.5*dt*df[id].z/dx[id].w;
		
		float mykin = 0.5*sep_cuda_dot(vel)*dx[id].w;
		float mymom = (dv[id].x + dv[id].y + dv[id].z)*dx[id].w;
		
		atomicAdd(&sums.x, mykin);
		atomicAdd(&sums.y, epot[id]);
		atomicAdd(&sums.z, mymom);
	}

	__syncthreads();
	
	if ( id < npart && threadIdx.x == 0 ) {
		atomicAdd(&(totalsum->x), sums.x);
		atomicAdd(&(totalsum->y), sums.y);
		atomicAdd(&(totalsum->z), sums.z);
	}
	
}

__global__ void oct_sep_cuda_sum_ekin(float3 *totalsum, const char type, float4* dx, float4 *dv, float4 *df, 
									  float dt, float *epot, unsigned npart){

	int id = blockIdx.x*blockDim.x + threadIdx.x;
	__shared__ float sumekin;
	__shared__ int numtype;

	if ( threadIdx.x==0 ){
	   	sumekin = 0.0f;
		numtype = 0;
	}
	__syncthreads();

	int itype = __float2int_rd(df[id].w);

	if ( id < npart && itype==(int)type ){
		float4 vel; 
		vel.x =  dv[id].x - 0.5*dt*df[id].x/dx[id].w;
		vel.y =  dv[id].y - 0.5*dt*df[id].y/dx[id].w;
		vel.z =  dv[id].z - 0.5*dt*df[id].z/dx[id].w;
		
		float myekin = 0.5*sep_cuda_dot(vel)*dx[id].w;
		
		atomicAdd(&sumekin, myekin);
		atomicAdd(&numtype, 1.0);
	}

	__syncthreads();
	
	if ( id < npart && threadIdx.x == 0 ){ 
		atomicAdd(&(totalsum->x), sumekin);
		atomicAdd(&(totalsum->y), numtype);
	}
}


__global__ void oct_sep_cuda_reset_variable(float3 *a){

	a->x = a->y = a->z = 0.0f;

}



#endif

__inline__ __device__ float sep_cuda_wrap(float x, float lbox){
	
	if ( x > 0.5*lbox ) 
		x -= lbox;
	else if  ( x < -0.5*lbox ) 
		x += lbox;
	
	return x;
}

__inline__ __device__ float sep_cuda_periodic(float x, float lbox, int *crossing){
	
	if ( x > lbox ) {
		x -= lbox;  
		*crossing = *crossing + 1;
	}
	else if  ( x < 0 ) {
		x += lbox;
		*crossing = *crossing - 1;
	}
	
	return x;
}


__global__ void sep_cuda_leapfrog(float4 *pos, float4 *vel, 
		  float4 *force, float *dist, int3 *crossing, float dt, float3 lbox, unsigned npart){

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	// Local variables not even necessary for speed up	
	float4 mypos = make_float4(pos[i].x, pos[i].y, pos[i].z, pos[i].w);
	float4 myvel = make_float4(vel[i].x, vel[i].y, vel[i].z, vel[i].w);

	if ( i < npart ) {
		float imass = 1.0/mypos.w;
		
		myvel.x += force[i].x*imass*dt; 
		myvel.y += force[i].y*imass*dt;
		myvel.z += force[i].z*imass*dt;
		
		mypos.x += myvel.x*dt;
		mypos.x = sep_cuda_periodic(mypos.x, lbox.x, &(crossing[i].x));
		
		mypos.y += myvel.y*dt;
		mypos.y = sep_cuda_periodic(mypos.y, lbox.y, &(crossing[i].y));
	
		mypos.z += myvel.z*dt;
		mypos.z = sep_cuda_periodic(mypos.z, lbox.z, &(crossing[i].z));
	
		pos[i].x = mypos.x; pos[i].y = mypos.y; pos[i].z = mypos.z;
		vel[i].x = myvel.x; vel[i].y = myvel.y; vel[i].z = myvel.z; 
	}
	
}


__global__ void sep_cuda_update_nosehoover(float *alpha, float3 *denergies, float temp0, 
										   float tau, float dt, unsigned int npart){

	float temp = (2.0/3.0)*denergies->x/npart; 

	*alpha = *alpha + dt/(tau*tau)*(temp/temp0 - 1.0);

}


__global__ void sep_cuda_update_nosehoover(float *alpha, float3 *denergies, float temp0, 
										   float tau, float dt){

	float temp = (2.0/3.0)*denergies->x/denergies->y; 

	*alpha = *alpha + dt/(tau*tau)*(temp/temp0 - 1.0);

}


__global__ void sep_cuda_nosehoover(float *alpha, float4 *pos, float4 *vel, float4 *force, unsigned npart){
	
	unsigned id = blockIdx.x*blockDim.x + threadIdx.x;

	if ( id < npart ){	
		float fac = (*alpha)*pos[id].w;
		force[id].x -= fac*vel[id].x; 
		force[id].y -= fac*vel[id].y; 
		force[id].z -= fac*vel[id].z;		
	}
}

__global__ void sep_cuda_nosehoover(float *alpha, const char type, float4 *pos, float4 *vel, float4 *force, unsigned npart){
	
	unsigned id = blockIdx.x*blockDim.x + threadIdx.x;
	int itype = __float2int_rd(force[id].w);

	if ( id < npart && itype == (int)type ){	
		float fac = (*alpha)*pos[id].w;
		force[id].x -= fac*vel[id].x; 
		force[id].y -= fac*vel[id].y; 
		force[id].z -= fac*vel[id].z;		
	}
}


void sep_cuda_thermostat_nh(sepcupart *pptr, float temp0, float tau){
	const int nb = pptr->sptr->nblocks; 
	const int nt = pptr->sptr->nthreads;
	
	// Get current system kinetic energy
#ifdef OCTAVE
	oct_sep_cuda_sumenergies<<<nb,nt>>>
		(pptr->sptr->denergies, pptr->dx, pptr->dv, pptr->df, pptr->sptr->dt, pptr->epot, pptr->sptr->npart);
#else 
	sep_cuda_sum_energies<<<nb,nt>>>
		(pptr->sptr->denergies, pptr->dx, pptr->dv, pptr->df, pptr->sptr->dt, pptr->epot, pptr->sptr->npart);
#endif

	//hipDeviceSynchronize();
	
	// Update nh-alpha dynamics (single thread)
	sep_cuda_update_nosehoover<<<1,1>>>
		(pptr->sptr->dalpha, pptr->sptr->denergies, temp0, tau, pptr->sptr->dt, pptr->sptr->npart);
	//hipDeviceSynchronize();

	// Add thermostat force
	sep_cuda_nosehoover<<<nb, nt>>>
		(pptr->sptr->dalpha, pptr->dx, pptr->dv, pptr->df, pptr->sptr->npart);
	//hipDeviceSynchronize();		

	sep_cuda_reset_variable<<<1,1>>>(pptr->sptr->denergies);	
}

 
void sep_cuda_thermostat_nh(const char type, sepcupart *pptr, float temp0, float tau){
	const int nb = pptr->sptr->nblocks; 
	const int nt = pptr->sptr->nthreads;
	
	// Get current system kinetic energy
#ifdef OCTAVE
	oct_sep_cuda_sum_ekin<<<nb,nt>>>
		(pptr->sptr->denergies, type, pptr->dx, pptr->dv, pptr->df, pptr->sptr->dt, pptr->epot, pptr->sptr->npart);
#else 
	sep_cuda_sum_ekin<<<nb,nt>>>
		(pptr->sptr->denergies, type, pptr->dx, pptr->dv, pptr->df, pptr->sptr->dt, pptr->epot, pptr->sptr->npart);
#endif

	//hipDeviceSynchronize();
	
	// Update nh-alpha dynamics (single thread)
	sep_cuda_update_nosehoover<<<1,1>>>
		(pptr->sptr->dalpha, pptr->sptr->denergies, temp0, tau, pptr->sptr->dt);
	//hipDeviceSynchronize();

	// Add thermostat force
	sep_cuda_nosehoover<<<nb, nt>>>
		(pptr->sptr->dalpha, type,  pptr->dx, pptr->dv, pptr->df, pptr->sptr->npart);
	hipDeviceSynchronize();		

#ifdef OCTAVE
	oct_sep_cuda_reset_variable<<<1,1>>>(pptr->sptr->denergies);
#else
	sep_cuda_reset_variable<<<1,1>>>(pptr->sptr->denergies);	
#endif

}

  
void sep_cuda_integrate_leapfrog(sepcupart *pptr){
	const int nb = pptr->sptr->nblocks; 
	const int nt = pptr->sptr->nthreads;

	sep_cuda_leapfrog<<<nb, nt>>>
		(pptr->dx, pptr->dv, pptr->df, pptr->ddist, pptr->dcrossings, pptr->sptr->dt, pptr->lbox, pptr->npart);
	//hipDeviceSynchronize();
	
}


