#include "hip/hip_runtime.h"

#include "sepcudamem.h"


sepcupart* sep_cuda_allocate_memory(unsigned npartPadding){
	sepcupart* ptr;
	
	if ( hipHostMalloc((void **)&ptr, sizeof(sepcupart))== hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	size_t nbytes = npartPadding*sizeof(float4);

	// Host
	if ( hipHostMalloc((void **)&(ptr->hx), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hv), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hf), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hx0), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->ht), npartPadding*sizeof(char)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hcrossings), npartPadding*sizeof(int3)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipHostMalloc((void **)&(ptr->hmolindex), npartPadding*sizeof(int)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	// Device
	if ( hipMalloc((void **)&(ptr->dx), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->dv), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->df), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->dx0), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->dxprev), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->ddist), npartPadding*sizeof(float)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->epot), npartPadding*sizeof(float)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->press), nbytes) == hipErrorOutOfMemory )
		sep_cuda_mem_error();

	if ( hipMalloc((void **)&(ptr->sumpress), sizeof(float4)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	ptr->maxneighb = SEP_CUDA_MAXNEIGHBS;
	if ( hipMalloc(&(ptr->neighblist), sizeof(int)*npartPadding*(ptr->maxneighb)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();

	if ( hipMalloc((void **)&(ptr->dcrossings), npartPadding*sizeof(int3)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(ptr->dmolindex), npartPadding*sizeof(int)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	return ptr;
}

void sep_cuda_free_memory(sepcupart *ptr){
		
	// System structure
	hipFree(ptr->sptr->denergies); hipHostFree(ptr->sptr->henergies);
	hipFree(ptr->sptr->dalpha); 
	
	hipHostFree(ptr->sptr);

	// Particle structure
	hipHostFree(ptr->hx); 	hipHostFree(ptr->hv); 
	hipHostFree(ptr->hf); 	hipHostFree(ptr->hx0);
	hipHostFree(ptr->ht);
	
   	hipHostFree(ptr->hcrossings); hipHostFree(ptr->hmolindex); 
	
	hipFree(ptr->dx); hipFree(ptr->dv); hipFree(ptr->df); hipFree(ptr->dx0);
	hipFree(ptr->ddist); hipFree(ptr->neighblist);
	hipFree(ptr->epot); hipFree(ptr->press); hipFree(ptr->sumpress); 
	
	hipFree(ptr->dcrossings); hipFree(ptr->dmolindex);
	
	hipHostFree(ptr);
}


sepcupart* sep_cuda_load_xyz(const char *xyzfile){
	unsigned npart;
	int nthreads = SEP_CUDA_NTHREADS;
	
	FILE *fin = fopen(xyzfile, "r");
	if ( fin == NULL )
		sep_cuda_file_error();
	
	fscanf(fin, "%d\n", &npart);
	
	unsigned nblocks = (npart + nthreads - 1) / nthreads;
	unsigned npartwithPadding = nblocks*nthreads;
	
	sepcupart *ptr = sep_cuda_allocate_memory(npartwithPadding);
		
	ptr->nblocks = nblocks; 
	ptr->nthreads = nthreads;
	ptr->npart = npart; 
	ptr->npart_padding = npartwithPadding;
	ptr->hexclusion_rule = SEP_CUDA_EXCL_NONE;
	ptr->sptr = NULL;

	fscanf(fin, "%f %f %f\n", &(ptr->lbox.x), &(ptr->lbox.y), &(ptr->lbox.z));

	// type, position, velocity, mass, charge	
	for ( unsigned n=0; n<npart; n++ ) {
		fscanf(fin, "%c %f %f %f %f %f %f %f %f\n", 
			   &(ptr->ht[n]), &(ptr->hx[n].x),&(ptr->hx[n].y),&(ptr->hx[n].z), 
			   &(ptr->hv[n].x),&(ptr->hv[n].y),&(ptr->hv[n].z), &(ptr->hx[n].w), &(ptr->hv[n].w));
		ptr->hcrossings[n].x = ptr->hcrossings[n].y = ptr->hcrossings[n].z = 0;
		ptr->hmolindex[n] = -1;
	}
	
	fclose(fin);

	for ( unsigned n=npart; n<npartwithPadding; n++ ){
		ptr->hx[n].x = ptr->hx[n].y = ptr->hx[n].z = 0.0f;
		ptr->hv[n].x = ptr->hv[n].y = ptr->hv[n].z = 0.0f;
		ptr->hv[n].w = 1.0; ptr->ht[n] = 'A';
		ptr->hmolindex[n] = -1;
	}
	
	sep_cuda_copy(ptr, 'x', 'd'); 
	sep_cuda_copy(ptr, 'v', 'd');
	sep_cuda_copy(ptr, 'f', 'd');
	sep_cuda_copy(ptr, 'c', 'd');
	
	hipMemcpy(ptr->dmolindex, ptr->hmolindex, npartwithPadding*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(ptr->dxprev, ptr->hx, npartwithPadding*sizeof(float4), hipMemcpyHostToDevice);

	return ptr;
}


sepcusys *sep_cuda_sys_setup(sepcupart *pptr){
	
	sepcusys *sptr;
	if ( hipHostMalloc((void **)&sptr, sizeof(sepcusys)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	sptr->npart = pptr->npart;
	sptr->npart_padding = pptr->npart_padding;
	
	sptr->nblocks = pptr->nblocks;
	sptr->nthreads = pptr->nthreads;
	
	sptr->dt = 0.005;
	sptr->skin = 0.3;
	sptr->lbox = pptr->lbox;
	sptr->iteration = -1;
	sptr->neighbupdate = true;	

	sptr->molprop = false;
	sptr->molpropinterval = 0;

	sptr->cmflag = false;

	sptr->pptr = pptr; sptr->mptr = NULL;
	pptr->sptr = sptr;

	if ( hipHostMalloc((void **)&(sptr->henergies), sizeof(float3)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(sptr->denergies), sizeof(float3)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	if ( hipMalloc((void **)&(sptr->dalpha), sizeof(float)) == hipErrorOutOfMemory )
		sep_cuda_mem_error();
	
	sep_cuda_set_value<<<1,1>>>(sptr->dalpha, 0.2);
	
	return sptr;
}


